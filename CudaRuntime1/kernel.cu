﻿#include <iostream>
#include <thread>
#include <cassert>
#include <algorithm>
#include <vector>
#include <iomanip> 

#include "hip/hip_runtime.h"


#define NumRunner 100
#define Distance 100

struct Runner {
    int ID;
    int Speed;
    int Position;

    Runner() {
        ID = 0;
        Speed = rand() % 5 + 1;
        Position = 0;
    }
};

// comparison operator for Runner struct
inline bool operator >(const Runner& lhs, const Runner& rhs)
{
    return lhs.Position > rhs.Position;
}

void PrintPositions(Runner runners[NumRunner]) {
    // array is sorted before printing so that the
    // result is in rank order
    std::sort(runners, runners + NumRunner, std::greater<Runner>());

    for (int i = 0; i < NumRunner; i++) {
        std::cout << "\n";
        std::cout << std::setw(3) << runners[i].ID << ": " << runners[i].Position;
    }
    std::cout << "\n\n";
}

__global__ void CalculateDisplacement(Runner runners[], bool* hasFinishedRace) {
   
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    // prevent invoking more threads than number of runners
    if (index < NumRunner) {
        // make runners run     
        for (int i = index; i < NumRunner; i += stride) {
            runners[index].Position += runners[index].Speed;

            if (runners[index].Position >= Distance) {
                *hasFinishedRace = true;
            }
        }
    }
}

int main()
{
    // allocate memory for runners
    Runner* runners; 
    hipMallocManaged(&runners, NumRunner * sizeof(Runner));

    // allocate memory for race finish condition
    bool* hasFinishedRace = new bool(false);
    hipMallocManaged(&hasFinishedRace, sizeof(bool));

    
    std::srand(std::time(nullptr)); // use current time as seed for random generator

    // initialize runners
    for (int i = 0; i < NumRunner; ++i) {
        runners[i] = Runner();
        runners[i].ID = i;
    }

    // set total thread amount
    const int numBlocks = 1;
    const int numThreads = 128;
    assert(numThreads % 32 == 0);
    assert(numBlocks * numThreads > NumRunner);

    while (!*hasFinishedRace) {
        CalculateDisplacement <<<numBlocks, numThreads>>> (runners, hasFinishedRace);
        hipDeviceSynchronize();
        std::this_thread::sleep_for(std::chrono::seconds(1));
        PrintPositions(runners);
    }   

    hipFree(runners);
    hipFree(hasFinishedRace);
    
}

